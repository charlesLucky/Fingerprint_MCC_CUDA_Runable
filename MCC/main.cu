#include <vector>
#include <iostream>
#include <algorithm>

#include <dirent.h>

#include "errors.h"
#include "debug.h"
#include "constants.cuh"
#include "template.cuh"
#include "matcher.cuh"
#include "io.cuh"
#include "mcc.cuh"
#include "consolidation.cuh"
#include <string>  

using namespace std;

bool buildTemplateFromFile(
    const char *input,
    const char *output) {
  int width, height, dpi, n;
  vector<Minutia> minutiae;
  if (!loadMinutiaeFromFile(input, width, height, dpi, n, minutiae))
    return false;

  vector<char> cylinderValidities, cellValidities, cellValues;
  buildTemplate(minutiae, width, height,
    cylinderValidities, cellValidities, cellValues);
  handleError(hipDeviceSynchronize());

  return saveTemplateToFile(
    output, width, height, dpi, n, minutiae,
    cylinderValidities.size(), cylinderValidities, cellValidities, cellValues);
}

bool buildSimilarityFromTemplate(
    const char *template1,
    const char *template2,
    const char *output) {
  int width1, height1, dpi1, n1;
  vector<Minutia> minutiae1;
  int m1;
  vector<char> cylinderValidities1, cellValidities1, cellValues1;
  if (!loadTemplateFromFile(template1,
      width1, height1, dpi1, n1, minutiae1,
      m1, cylinderValidities1, cellValidities1, cellValues1))
    return false;

  int width2, height2, dpi2, n2;
  vector<Minutia> minutiae2;
  int m2;
  vector<char> cylinderValidities2, cellValidities2, cellValues2;
  if (!loadTemplateFromFile(template2,
      width2, height2, dpi2, n2, minutiae2,
      m2, cylinderValidities2, cellValidities2, cellValues2))
    return false;

  vector<float> matrix;
  matchTemplate(
    minutiae1, cylinderValidities1, cellValidities1, cellValues1,
    minutiae2, cylinderValidities2, cellValidities2, cellValues2,
    matrix);
  auto similarity = LSSR(matrix, m1, m2, minutiae1, minutiae2);
  printf("Similarity: %f\n", similarity);
  return saveSimilarityToFile(output, m1, m2, matrix);
}


float buildSimilarityFromTemplateV2(
	const char *template1,
	const char *template2) {
	int width1, height1, dpi1, n1;
	vector<Minutia> minutiae1;
	int m1;
	vector<char> cylinderValidities1, cellValidities1, cellValues1;
	if (!loadTemplateFromFile(template1,
		width1, height1, dpi1, n1, minutiae1,
		m1, cylinderValidities1, cellValidities1, cellValues1))
		return false;

	int width2, height2, dpi2, n2;
	vector<Minutia> minutiae2;
	int m2;
	vector<char> cylinderValidities2, cellValidities2, cellValues2;
	if (!loadTemplateFromFile(template2,
		width2, height2, dpi2, n2, minutiae2,
		m2, cylinderValidities2, cellValidities2, cellValues2))
		return false;

	vector<float> matrix;
	matchTemplate(
		minutiae1, cylinderValidities1, cellValidities1, cellValues1,
		minutiae2, cylinderValidities2, cellValidities2, cellValues2,
		matrix);
	auto similarity = LSSR(matrix, m1, m2, minutiae1, minutiae2);
	printf("Similarity: %f\n", similarity);
	return similarity;
}

bool buildSimilarityFromMinutiae(
    const char *minutiae1,
    const char *minutiae2,
    const char *output) {
  MCC mcc(minutiae1);
  if (!mcc.load() || !mcc.build()) return false;

  float similarity;
  int n, m;
  vector<float> matrix;
  bool ret = mcc.match(minutiae2, similarity, n, m, matrix);
  if (!ret) return false;
  printf("Similarity: %f\n", similarity);
  return saveSimilarityToFile(output, n, m, matrix);
}

bool matchMany(const char *input, const char *targetDir) {
  DIR *dir;
  struct dirent *ent;
  vector<string> targets;
  vector<float> values;
  string stargetDir(targetDir);
  if (stargetDir.back() != '/')
    stargetDir += '/';

  if ((dir = opendir(targetDir)) != NULL) {
    while ((ent = readdir(dir)) != NULL) {
      if (ent->d_type != DT_REG)
        continue;
      targets.push_back(stargetDir + string(ent->d_name));
    }
    closedir(dir);
    values.resize(targets.size());
    MCC mcc(input, false);
    mcc.matchMany(targets, values);
    return true;
  }
  return false;
}

void printUsage(char const *argv[]) {
  cerr << "usage: " << argv[0] << " [mcc|template|match] [options]\n";
  cerr << endl;
  cerr << "mcc\t\t: <in:minutia1> <in:minutia2> <out:similarity>\n";
  cerr << "template\t: <in:minutia> <out:template>\n";
  cerr << "match\t\t: <in:template1> <in:template2> <out:similarity>\n";
  cerr << "many\t\t: <in:minutia> <in:dir>\n";
}


void comb(int N, int K)
{
	std::string bitmask(K, 1); // K leading 1's
	bitmask.resize(N, 0); // N-K trailing 0's
						  // print integers and permute bitmask
	do {
		for (int i = 0; i < N; ++i) // [0..N-1] integers
		{
			if (bitmask[i]) std::cout << " " << i;
		}
		std::cout << std::endl;
	} while (std::prev_permutation(bitmask.begin(), bitmask.end()));
}

int main(int argc, char const *argv[]) {
	/*
 if (argc > 1) {
    if (strncmp(argv[1], "mcc", 3) == 0 && argc == 5) {
      return !buildSimilarityFromMinutiae(argv[2], argv[3], argv[4]);
    } else if (strncmp(argv[1], "template", 8) == 0 && argc == 4) {
      return !buildTemplateFromFile(argv[2], argv[3]);
    } else if (strncmp(argv[1], "match", 5) == 0 && argc == 5) {
      return !buildSimilarityFromTemplate(argv[2], argv[3], argv[4]);
    } else if (strncmp(argv[1], "many", 4) == 0 && argc == 4) {
      return !matchMany(argv[2], argv[3]);
    }
  }
  printUsage(argv);
  */
	 
  /*
	string pointsdir = "C:/project source file/SampleMinutiae";
	string outdir = "C:/project source file/SampleMinutiae_OUT";
	for (int i = 1; i <= 10; i++) {
		for (int j = 1; j <= 12; j++) {
			string a = pointsdir + "/" + to_string(i) + "_" + to_string(j) + ".txt";
			string b = outdir + "/" + to_string(i) + "_" + to_string(j) + ".txt";
			buildTemplateFromFile(a.c_str(), b.c_str());
		}
	} 
	*/

	string pointsdir = "C:/project source file/SampleMinutiae";
	string outdir = "C:/project source file/SampleMinutiae_OUT";
	float gen[70];
	int count = 0;
	for (int i = 1; i <= 10; i++) {
		for (int j = 2; j <= 8; j++) {
			string a = outdir + "/" + to_string(i) + "_" + to_string(1) + ".txt";
			string b = outdir + "/" + to_string(i) + "_" + to_string(j) + ".txt";

			float simi = buildSimilarityFromTemplateV2(a.c_str(), b.c_str());
		}
	}
	std::cout << "**********************" << std::endl;


for (int i = 1; i < 10; i++) {
		
	for (int j = i+1; j <= 10; j++) {
		string a = outdir + "/" + to_string(i) + "_" + to_string(1) + ".txt";
		string b = outdir + "/" + to_string(j) + "_" + to_string(1) + ".txt";
		float simi = buildSimilarityFromTemplateV2(a.c_str(), b.c_str());
		 
	}
} ;

	return 1;

}
