#include "hip/hip_runtime.h"
#include "matcher.cuh"
#include "binarization.cuh"
#include "minutia.cuh"
#include "constants.cuh"
#include "util.cuh"
#include "errors.h"
#include "debug.h"

#include "hip/hip_runtime.h"
#include ""

using namespace std;

__global__
void computeSimilarity(
    Minutia *minutiae1,
    char *cylinderValidities1,
    unsigned int *binarizedValidities1,
    unsigned int *binarizedValues1,
    Minutia *minutiae2,
    char *cylinderValidities2,
    unsigned int *binarizedValidities2,
    unsigned int *binarizedValues2,
    float *matrix, int rows, int cols) {
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  if (row >= rows || col >= cols) return;

  if (!cylinderValidities1[row] || !cylinderValidities2[col] ||
      floatGreater(
        fabsf(angle(minutiae1[row].theta, minutiae2[col].theta)),DELTA_THETA)) {
    matrix[row*cols + col] = 0.0f;
    return;
  }

  int intPerCylinder = NC/BITS;
  int rowIdx = row * intPerCylinder;
  int colIdx = col * intPerCylinder;

  int maskBits = 0, rowBits = 0, colBits = 0, xorBits = 0;
  for (int i = 0; i < intPerCylinder; ++i) {
    auto mask = binarizedValidities1[rowIdx+i] & binarizedValidities2[colIdx+i];
    auto rowValue = binarizedValues1[rowIdx+i] & mask;
    auto colValue = binarizedValues2[colIdx+i] & mask;
    auto xorValue = rowValue ^ colValue;
    maskBits += __popc(mask);
    rowBits += __popc(rowValue);
    colBits += __popc(colValue);
    xorBits += __popc(xorValue);
  }

  bool matchable = maskBits >= MIN_ME_CELLS && (rowBits || colBits);
  float similarity = matchable
    ? (1 - sqrtf(xorBits) / (sqrtf(rowBits)+sqrtf(colBits)))
    : 0.0f;
  matrix[row*cols + col] = similarity;
}

__host__
void devMatchTemplate(
    Minutia *devMinutiae1, const int n,
    char *devCylinderValidities1,
    unsigned int *devBinarizedValidities1,
    unsigned int *devBinarizedValues1,
    Minutia *devMinutiae2, const int m,
    char *devCylinderValidities2,
    unsigned int *devBinarizedValidities2,
    unsigned int *devBinarizedValues2,
    float *devMatrix) {
  int threadPerDim = 32;
  dim3 blockCount(ceilMod(m, threadPerDim), ceilMod(n, threadPerDim));
  dim3 threadCount(threadPerDim, threadPerDim);
  computeSimilarity<<<blockCount, threadCount>>>(
    devMinutiae1, devCylinderValidities1, devBinarizedValidities1, devBinarizedValues1,
    devMinutiae2, devCylinderValidities2, devBinarizedValidities2, devBinarizedValues2,
    devMatrix, n, m);
  handleError(
    hipPeekAtLastError());
}

__host__
void matchTemplate(
    const vector<Minutia>& minutiae1,
    const vector<char>& cylinderValidities1,
    const vector<char>& cellValidities1,
    const vector<char>& cellValues1,
    const vector<Minutia>& minutiae2,
    const vector<char>& cylinderValidities2,
    const vector<char>& cellValidities2,
    const vector<char>& cellValues2,
    vector<float>& matrix) {

  Minutia *devMinutiae1, *devMinutiae2;
  char *devCylinderValidities1, *devCylinderValidities2;
  char *devCellValidities1, *devCellValidities2;
  char *devCellValues1, *devCellValues2;
  size_t devMinutiae1Size = minutiae1.size() * sizeof(Minutia);
  size_t devMinutiae2Size = minutiae2.size() * sizeof(Minutia);
  size_t devCylinderValidities1Size = cylinderValidities1.size() * sizeof(char);
  size_t devCylinderValidities2Size = cylinderValidities2.size() * sizeof(char);
  size_t devCellValidities1Size = cellValidities1.size()  * sizeof(char);
  size_t devCellValidities2Size = cellValidities2.size()  * sizeof(char);
  size_t devCellValues1Size = cellValues1.size() * sizeof(char);
  size_t devCellValues2Size = cellValues2.size() * sizeof(char);
  handleError(
    hipMalloc(&devMinutiae1, devMinutiae1Size));
  handleError(
    hipMemcpy(devMinutiae1, minutiae1.data(), devMinutiae1Size, hipMemcpyHostToDevice));
  handleError(
    hipMalloc(&devMinutiae2, devMinutiae2Size));
  handleError(
    hipMemcpy(devMinutiae2, minutiae2.data(), devMinutiae2Size, hipMemcpyHostToDevice));
  handleError(
    hipMalloc(&devCylinderValidities1, devCylinderValidities1Size));
  handleError(
    hipMemcpy(devCylinderValidities1, cylinderValidities1.data(), devCylinderValidities1Size, hipMemcpyHostToDevice));
  handleError(
    hipMalloc(&devCylinderValidities2, devCylinderValidities2Size));
  handleError(
    hipMemcpy(devCylinderValidities2, cylinderValidities2.data(), devCylinderValidities2Size, hipMemcpyHostToDevice));
  handleError(
    hipMalloc(&devCellValidities1, devCellValidities1Size));
  handleError(
    hipMemcpy(devCellValidities1, cellValidities1.data(), devCellValidities1Size, hipMemcpyHostToDevice));
  handleError(
    hipMalloc(&devCellValidities2, devCellValidities2Size));
  handleError(
    hipMemcpy(devCellValidities2, cellValidities2.data(), devCellValidities2Size, hipMemcpyHostToDevice));
  handleError(
    hipMalloc(&devCellValues1, devCellValues1Size));
  handleError(
    hipMemcpy(devCellValues1, cellValues1.data(), devCellValues1Size, hipMemcpyHostToDevice));
  handleError(
    hipMalloc(&devCellValues2, devCellValues2Size));
  handleError(
    hipMemcpy(devCellValues2, cellValues2.data(), devCellValues2Size, hipMemcpyHostToDevice));

  int intPerCylinder = NC/BITS;
  unsigned int *devBinarizedValidities1, *devBinarizedValues1;
  unsigned int *devBinarizedValidities2, *devBinarizedValues2;
  size_t devBinarizedValidities1Size = minutiae1.size() * intPerCylinder * sizeof(unsigned int);
  size_t devBinarizedValues1Size = minutiae1.size() * intPerCylinder * sizeof(unsigned int);
  size_t devBinarizedValidities2Size = minutiae2.size() * intPerCylinder * sizeof(unsigned int);
  size_t devBinarizedValues2Size = minutiae2.size() * intPerCylinder * sizeof(unsigned int);
  handleError(
    hipMalloc(&devBinarizedValidities1, devBinarizedValidities1Size));
  handleError(
    hipMalloc(&devBinarizedValidities2, devBinarizedValidities2Size));
  handleError(
    hipMalloc(&devBinarizedValues1, devBinarizedValues1Size));
  handleError(
    hipMalloc(&devBinarizedValues2, devBinarizedValues2Size));

  devBinarizedTemplate(minutiae1.size(),
    devCellValidities1, devCellValues1,
    devBinarizedValidities1, devBinarizedValues1);
  devBinarizedTemplate(minutiae2.size(),
    devCellValidities2, devCellValues2,
    devBinarizedValidities2, devBinarizedValues2);

  float *devMatrix;
  size_t devMatrixSize = minutiae1.size() * minutiae2.size() * sizeof(float);
  handleError(
    hipMalloc(&devMatrix, devMatrixSize));

  devMatchTemplate(
    devMinutiae1, minutiae1.size(),
    devCylinderValidities1, devBinarizedValidities1, devBinarizedValues1,
    devMinutiae2, minutiae2.size(),
    devCylinderValidities2, devBinarizedValidities2, devBinarizedValues2,
    devMatrix);

  matrix.resize(minutiae1.size() * minutiae2.size());
  handleError(
    hipMemcpy(matrix.data(), devMatrix, devMatrixSize, hipMemcpyDeviceToHost));

  hipFree(devMinutiae1);
  hipFree(devMinutiae2);
  hipFree(devCylinderValidities1);
  hipFree(devCylinderValidities2);
  hipFree(devCellValidities1);
  hipFree(devCellValidities2);
  hipFree(devCellValues1);
  hipFree(devCellValues2);
  hipFree(devBinarizedValidities1);
  hipFree(devBinarizedValidities2);
  hipFree(devBinarizedValues1);
  hipFree(devBinarizedValues2);
  hipFree(devMatrix);
}
