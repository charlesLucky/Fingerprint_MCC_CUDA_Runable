#include "hip/hip_runtime.h"
#include <vector>

#include "minutia.cuh"
#include "area.cuh"
#include "constants.cuh"
#include "util.cuh"
#include "errors.h"
#include "debug.h"

#include "hip/hip_runtime.h"
#include ""

using namespace std;

bool initialized = false;
int numCellsInCylinder = 0;

__host__ void initialize() {
  if (initialized) return;
  initialized = true;

  numCellsInCylinder = 0;
  float temp = DELTA_S/2;
  for (int i = 0; i < NS; ++i) {
    float x = DELTA_S * i + temp;
    float dx = x-R;
    for (int j = 0; j < NS; ++j) {
      float y = DELTA_S * j + temp;
      float dy = y-R;
      if (dx*dx + dy*dy <= R_SQR) ++numCellsInCylinder;
    }
  }
}

__host__ __device__ __inline__
float spatialContribution(
    int mt_x, int mt_y, int pi, int pj) {
  auto gaussian = [&](int t_sqr) -> float {
    return I_2_SIGMA_S_SQRT_PI * expf(-t_sqr * I_2_SIGMA_S_SQR);
  };
  return gaussian(sqrDistance(mt_x, mt_y, pi, pj));
}

__host__ __device__ __inline__
float directionalContribution(
    float m_theta, float mt_theta, float dphik) {
  // http://www.wolframalpha.com/input/?i=integrate+(e%5E(-(t%5E2)%2F(2(x%5E2)))+dt)
  auto integrate = [&](float val) -> float {
    return SQRT_PI_2_SIGMA_D * erff(val * I_SQRT_2_SIGMA_D);
  };
  auto gaussian = [&](float val) -> float {
    return I_SQRT_2_PI_SIGMA_D *
      (integrate(val+DELTA_D_2)-integrate(val-DELTA_D_2));
  };
  return gaussian(
    angle(dphik, angle(m_theta, mt_theta)));
}

__global__
void buildCylinder(
    Minutia *minutiae,
    int width, int height,
    char *validArea,
    int numCellsInCylinder,
    char *cylinderValidities,
    char *cellValidities,
    char *cellValues) {
  extern __shared__ int shared[];

  const int N = gridDim.x;
  Minutia *sharedMinutiae = (Minutia*)shared;

  int idxMinutia = blockIdx.x;
  int idxThread = threadIdx.y * blockDim.x + threadIdx.x;
  int contributed = 0;

  if (idxThread < N) {
    sharedMinutiae[idxThread] = minutiae[idxThread];
    if (idxThread != idxMinutia) {
      auto dist = sqrDistance(
        sharedMinutiae[idxThread].x, sharedMinutiae[idxThread].y,
        minutiae[idxMinutia].x, minutiae[idxMinutia].y);
      contributed = dist <= (R+SIGMA_3S)*(R+SIGMA_3S);
    }
  }
  int sumContributed = __syncthreads_count(contributed);

  Minutia m = sharedMinutiae[idxMinutia];

  float halfNS = (NS + 1) / 2.0f;
  float halfNSi = (threadIdx.x+1) - halfNS;
  float halfNSj = (threadIdx.y+1) - halfNS;
  float sint, cost;
  sincosf(m.theta, &sint, &cost);
  int pi = m.x + roundf(DELTA_S * (cost * halfNSi + sint * halfNSj));
  int pj = m.y + roundf(DELTA_S * (-sint * halfNSi + cost * halfNSj));

  char validity = pi >= 0 && pi < width && pj >= 0 && pj < height
    && validArea[pj * width + pi]
    && sqrDistance(m.x, m.y, pi, pj) <= R_SQR;

  int idx = idxMinutia * NC + threadIdx.y * NS * ND + threadIdx.x * ND;
  for (int k = 0; k < ND; ++k) {
    char value = 0;

    if (validity) {
      float dphik = -M_PI + (k + 0.5f) * DELTA_D;
      float sum = 0.0f;

      for (int l = 0; l < N; ++l) {
        if (l == idxMinutia)
          continue;

        Minutia mt(sharedMinutiae[l]);
        if (sqrDistance(mt.x, mt.y, pi, pj) > SIGMA_9S_SQR)
          continue;

        float sContrib = spatialContribution(mt.x, mt.y, pi, pj);
        float dContrib = directionalContribution(m.theta, mt.theta, dphik);
        sum += sContrib * dContrib;
      }

      if (sum >= MU_PSI)
        value = 1;
    }
    cellValidities[idx+k] = validity;
    cellValues[idx+k] = value;
  }

  int sumValidities = __syncthreads_count(validity);
  if (threadIdx.x == 0 && threadIdx.y == 0) {
    cylinderValidities[idxMinutia] = sumContributed >= MIN_M &&
      (float)sumValidities/(numCellsInCylinder) >= MIN_VC;
    devDebug("Minutia %2d VC: ((%3d/%d) = %.5f) >= %.2f, M: %2d >= %d\n",
      idxMinutia,
      sumValidities, numCellsInCylinder,
      (float)sumValidities/(numCellsInCylinder), MIN_VC,
      sumContributed, MIN_M);
  }
}

__host__
void devBuildTemplate(
    Minutia *devMinutiae, const int n,
    char *devArea, const int width, const int height,
    char *devCylinderValidities,
    char *devCellValidities,
    char *devCellValues) {

  initialize();

  dim3 blockDim(NS, NS);
  int sharedSize = n * sizeof(Minutia);
  buildCylinder<<<n, blockDim, sharedSize>>>(
    devMinutiae, width, height, devArea, numCellsInCylinder,
    devCylinderValidities, devCellValidities, devCellValues);
}

__host__
void buildTemplate(
    const vector<Minutia>& minutiae,
    const int width, const int height,
    vector<char>& cylinderValidities,
    vector<char>& cellValidities,
    vector<char>& cellValues) {

  auto area = buildValidArea(minutiae, width, height);

  Minutia *devMinutiae;
  char *devArea;
  char *devCylinderValidities, *devCellValidities, *devCellValues;
  size_t devMinutiaeSize = minutiae.size() * sizeof(Minutia);
  size_t devAreaSize = width * height * sizeof(char);
  size_t devCylinderValiditiesSize = minutiae.size() * sizeof(char);
  size_t devCellValiditiesSize = minutiae.size() * NC * sizeof(char);
  size_t devCellValuesSize = minutiae.size() * NC * sizeof(char);
  handleError(
    hipMalloc(&devMinutiae, devMinutiaeSize));
  handleError(
    hipMemcpy(devMinutiae, minutiae.data(), devMinutiaeSize, hipMemcpyHostToDevice));
  handleError(
    hipMalloc(&devArea, devAreaSize));
  handleError(
    hipMemcpy(devArea, area.data(), devAreaSize, hipMemcpyHostToDevice));
  handleError(
    hipMalloc(&devCylinderValidities, devCylinderValiditiesSize));
  handleError(
    hipMalloc(&devCellValues, devCellValuesSize));
  handleError(
    hipMalloc(&devCellValidities, devCellValiditiesSize));

  devBuildTemplate(
    devMinutiae, minutiae.size(),
    devArea, width, height,
    devCylinderValidities,
    devCellValidities,
    devCellValues);

  cylinderValidities.resize(minutiae.size());
  cellValidities.resize(minutiae.size() * NC);
  cellValues.resize(minutiae.size() * NC);
  handleError(
    hipMemcpy(cylinderValidities.data(), devCylinderValidities, devCylinderValiditiesSize, hipMemcpyDeviceToHost));
  handleError(
    hipMemcpy(cellValidities.data(), devCellValidities, devCellValiditiesSize, hipMemcpyDeviceToHost));
  handleError(
    hipMemcpy(cellValues.data(), devCellValues, devCellValuesSize, hipMemcpyDeviceToHost));

  hipFree(devMinutiae);
  hipFree(devArea);
  hipFree(devCylinderValidities);
  hipFree(devCellValidities);
  hipFree(devCellValues);
}
